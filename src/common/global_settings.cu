#include "hip/hip_runtime.h"
#include "pyroclastmpm/common/global_settings.cuh"

namespace pyroclastmpm
{

  char output_directory_cpu[256];

#ifdef CUDA_ENABLED
  __constant__ SFType shape_function_gpu = LinearShapeFunction;

  __constant__ int num_surround_nodes_gpu;
  __constant__ Real dt_gpu;
  __constant__ Real inv_cell_size_gpu;
  __constant__ int global_step_gpu = 0;
  __constant__ int window_size_gpu;
  __constant__ int particles_per_cell_gpu;
  __constant__ int forward_window_gpu[64][3];
  __constant__ int backward_window_gpu[64][3];
#endif

  SFType shape_function_cpu = LinearShapeFunction;
  int num_surround_nodes_cpu;
  Real dt_cpu;
  Real inv_cell_size_cpu;
  int global_step_cpu = 0;
  int window_size_cpu;

  int particles_per_cell_cpu;

  const int linear_forward_window_1d[64][3] = {{0, 0, 0}, {1, 0, 0}};

  const int linear_forward_window_2d[64][3] = {{0, 0, 0},
                                               {1, 0, 0},
                                               {0, 1, 0},
                                               {1, 1, 0}};

  const int linear_forward_window_3d[64][3] = {{0, 0, 0}, {0, 0, 1}, {1, 0, 0}, {1, 0, 1}, {0, 1, 0}, {0, 1, 1}, {1, 1, 0}, {1, 1, 1}};

  const int linear_backward_window_1d[64][3] = {{0, 0, 0}, {-1, 0, 0}};

  const int linear_backward_window_2d[64][3] = {{0, 0, 0},
                                                {-1, 0, 0},
                                                {0, -1, 0},
                                                {-1, -1, 0}};

  const int linear_backward_window_3d[64][3] = {
      {0, 0, 0}, {0, 0, -1}, {-1, 0, 0}, {-1, 0, -1}, {0, -1, 0}, {0, -1, -1}, {-1, -1, 0}, {-1, -1, -1}};

  const int quadratic_forward_window_1d[64][3] = {{-1, 0, 0},
                                                  {0, 0, 0},
                                                  {1, 0, 0},
                                                  {2, 0, 0}};

  const int quadratic_forward_window_2d[64][3] = {
      {-1, -1, 0}, {0, -1, 0}, {1, -1, 0}, {2, -1, 0}, {-1, 0, 0}, {0, 0, 0}, {1, 0, 0}, {2, 0, 0}, {-1, 1, 0}, {0, 1, 0}, {1, 1, 0}, {2, 1, 0}, {-1, 2, 0}, {0, 2, 0}, {1, 2, 0}, {2, 2, 0}};

  const int quadratic_forward_window_3d[64][3] = {
      {-1, -1, -1}, {-1, -1, 0}, {-1, -1, 1}, {-1, -1, 2}, {0, -1, -1}, {0, -1, 0}, {0, -1, 1}, {0, -1, 2}, {1, -1, -1}, {1, -1, 0}, {1, -1, 1}, {1, -1, 2}, {2, -1, -1}, {2, -1, 0}, {2, -1, 1}, {2, -1, 2}, {-1, 0, -1}, {-1, 0, 0}, {-1, 0, 1}, {-1, 0, 2}, {0, 0, -1}, {0, 0, 0}, {0, 0, 1}, {0, 0, 2}, {1, 0, -1}, {1, 0, 0}, {1, 0, 1}, {1, 0, 2}, {2, 0, -1}, {2, 0, 0}, {2, 0, 1}, {2, 0, 2}, {-1, 1, -1}, {-1, 1, 0}, {-1, 1, 1}, {-1, 1, 2}, {0, 1, -1}, {0, 1, 0}, {0, 1, 1}, {0, 1, 2}, {1, 1, -1}, {1, 1, 0}, {1, 1, 1}, {1, 1, 2}, {2, 1, -1}, {2, 1, 0}, {2, 1, 1}, {2, 1, 2}, {-1, 2, -1}, {-1, 2, 0}, {-1, 2, 1}, {-1, 2, 2}, {0, 2, -1}, {0, 2, 0}, {0, 2, 1}, {0, 2, 2}, {1, 2, -1}, {1, 2, 0}, {1, 2, 1}, {1, 2, 2}, {2, 2, -1}, {2, 2, 0}, {2, 2, 1}, {2, 2, 2}};

  const int quadratic_backward_window_1d[64][3] = {{1, 0, 0},
                                                   {0, 0, 0},
                                                   {-1, 0, 0},
                                                   {-2, 0, 0}};

  const int quadratic_backward_window_2d[64][3] = {
      {1, 1, 0}, {0, 1, 0}, {-1, 1, 0}, {-2, 1, 0}, {1, 0, 0}, {0, 0, 0}, {-1, 0, 0}, {-2, 0, 0}, {1, -1, 0}, {0, -1, 0}, {-1, -1, 0}, {-2, -1, 0}, {1, -2, 0}, {0, -2, 0}, {-1, -2, 0}, {-2, -2, 0}};

  const int quadratic_backward_window_3d[64][3] = {
      {1, 1, 1}, {1, 1, 0}, {1, 1, -1}, {1, 1, -2}, {0, 1, 1}, {0, 1, 0}, {0, 1, -1}, {0, 1, -2}, {-1, 1, 1}, {-1, 1, 0}, {-1, 1, -1}, {-1, 1, -2}, {-2, 1, 1}, {-2, 1, 0}, {-2, 1, -1}, {-2, 1, -2}, {1, 0, 1}, {1, 0, 0}, {1, 0, -1}, {1, 0, -2}, {0, 0, 1}, {0, 0, 0}, {0, 0, -1}, {0, 0, -2}, {-1, 0, 1}, {-1, 0, 0}, {-1, 0, -1}, {-1, 0, -2}, {-2, 0, 1}, {-2, 0, 0}, {-2, 0, -1}, {-2, 0, -2}, {1, -1, 1}, {1, -1, 0}, {1, -1, -1}, {1, -1, -2}, {0, -1, 1}, {0, -1, 0}, {0, -1, -1}, {0, -1, -2}, {-1, -1, 1}, {-1, -1, 0}, {-1, -1, -1}, {-1, -1, -2}, {-2, -1, 1}, {-2, -1, 0}, {-2, -1, -1}, {-2, -1, -2}, {1, -2, 1}, {1, -2, 0}, {1, -2, -1}, {1, -2, -2}, {0, -2, 1}, {0, -2, 0}, {0, -2, -1}, {0, -2, -2}, {-1, -2, 1}, {-1, -2, 0}, {-1, -2, -1}, {-1, -2, -2}, {-2, -2, 1}, {-2, -2, 0}, {-2, -2, -1}, {-2, -2, -2}};

  void set_globals(const Real _dt,
                   const int particles_per_cell,
                   SFType _shapefunction,
                   const std::string _output_dir)
  {
    set_global_dt(_dt);
    set_global_shapefunction(_shapefunction);
    set_global_output_directory(_output_dir);
    set_global_particles_per_cell(particles_per_cell);
  }

  void set_global_particles_per_cell(const int _particles_per_cell)
  {
    particles_per_cell_cpu = _particles_per_cell;
#ifdef CUDA_ENABLED
    hipMemcpyToSymbol(HIP_SYMBOL(particles_per_cell_gpu), &(_particles_per_cell), sizeof(int), 0);
#endif
  }

  void set_global_output_directory(const std::string _output_dir)
  {
    std::strcpy(output_directory_cpu, _output_dir.c_str());
  };

  void set_global_dt(const Real _dt)
  {
    dt_cpu = _dt;
#ifdef CUDA_ENABLED
    hipMemcpyToSymbol(HIP_SYMBOL(dt_gpu), &(_dt), sizeof(Real), 0);
#endif
  };

  void set_global_step(const int _step)
  {
    global_step_cpu = _step;
#ifdef CUDA_ENABLED
    hipMemcpyToSymbol(HIP_SYMBOL(global_step_gpu), &(_step), sizeof(int), 0);
#endif
  };

  void set_global_shapefunction(SFType _shapefunction)
  {
    // TODO this can be constant defined in the kernel
    if (_shapefunction == LinearShapeFunction)
    {
      window_size_cpu = 2;
      num_surround_nodes_cpu = pow(window_size_cpu, DIM);
      shape_function_cpu = LinearShapeFunction;
#ifdef CUDA_ENABLED
      hipMemcpyToSymbol(HIP_SYMBOL(shape_function_gpu), &(shape_function_cpu), sizeof(int), 0);
#if DIM == 1
      hipMemcpyToSymbol(HIP_SYMBOL(forward_window_gpu), linear_forward_window_1d,
                         64 * 3 * sizeof(int));
      hipMemcpyToSymbol(HIP_SYMBOL(backward_window_gpu), linear_backward_window_1d,
                         64 * 3 * sizeof(int));
#elif DIM == 2
      hipMemcpyToSymbol(HIP_SYMBOL(forward_window_gpu), linear_forward_window_2d,
                         64 * 3 * sizeof(int));
      hipMemcpyToSymbol(HIP_SYMBOL(backward_window_gpu), linear_backward_window_2d,
                         64 * 3 * sizeof(int));
#else
      hipMemcpyToSymbol(HIP_SYMBOL(forward_window_gpu), linear_forward_window_3d,
                         64 * 3 * sizeof(int));
      hipMemcpyToSymbol(HIP_SYMBOL(backward_window_gpu), linear_backward_window_3d,
                         64 * 3 * sizeof(int));
#endif
#endif
    }
    else if (_shapefunction == QuadraticShapeFunction)
    {
      window_size_cpu = 4;
      num_surround_nodes_cpu = pow(window_size_cpu, DIM);
      shape_function_cpu = QuadraticShapeFunction;
#ifdef CUDA_ENABLED
      hipMemcpyToSymbol(HIP_SYMBOL(shape_function_gpu), &(shape_function_cpu), sizeof(int),
                         0);

#if DIM == 1
      hipMemcpyToSymbol(HIP_SYMBOL(forward_window_gpu), quadratic_forward_window_1d,
                         64 * 3 * sizeof(int));
      hipMemcpyToSymbol(HIP_SYMBOL(backward_window_gpu), quadratic_backward_window_1d,
                         64 * 3 * sizeof(int));
#elif DIM == 2
      hipMemcpyToSymbol(HIP_SYMBOL(forward_window_gpu), quadratic_forward_window_2d,
                         64 * 3 * sizeof(int));
      hipMemcpyToSymbol(HIP_SYMBOL(backward_window_gpu), quadratic_backward_window_2d,
                         64 * 3 * sizeof(int));
#else
      hipMemcpyToSymbol(HIP_SYMBOL(forward_window_gpu), quadratic_forward_window_3d,
                         64 * 3 * sizeof(int));
      hipMemcpyToSymbol(HIP_SYMBOL(backward_window_gpu), quadratic_backward_window_3d,
                         64 * 3 * sizeof(int));
#endif
#endif
    }
    else if (_shapefunction == CubicShapeFunction)
    {
      window_size_cpu = 4;
      num_surround_nodes_cpu = pow(window_size_cpu, DIM);
      shape_function_cpu = CubicShapeFunction;

#ifdef CUDA_ENABLED
      hipMemcpyToSymbol(HIP_SYMBOL(shape_function_gpu), &(shape_function_cpu), sizeof(int),
                         0);
#if DIM == 1
      hipMemcpyToSymbol(HIP_SYMBOL(forward_window_gpu), quadratic_forward_window_1d,
                         64 * 3 * sizeof(int));
      hipMemcpyToSymbol(HIP_SYMBOL(backward_window_gpu), quadratic_backward_window_1d,
                         64 * 3 * sizeof(int));
#elif DIM == 2
      hipMemcpyToSymbol(HIP_SYMBOL(forward_window_gpu), quadratic_forward_window_2d,
                         64 * 3 * sizeof(int));
      hipMemcpyToSymbol(HIP_SYMBOL(backward_window_gpu), quadratic_backward_window_2d,
                         64 * 3 * sizeof(int));
#else
      hipMemcpyToSymbol(HIP_SYMBOL(forward_window_gpu), quadratic_forward_window_3d,
                         64 * 3 * sizeof(int));
      hipMemcpyToSymbol(HIP_SYMBOL(backward_window_gpu), quadratic_backward_window_3d,
                         64 * 3 * sizeof(int));
#endif

#endif
    }
#ifdef CUDA_ENABLED
    hipMemcpyToSymbol(HIP_SYMBOL(window_size_gpu), &(window_size_cpu), sizeof(int), 0);

    hipMemcpyToSymbol(HIP_SYMBOL(num_surround_nodes_gpu), &(num_surround_nodes_cpu),
                       sizeof(int), 0);
#endif
  };



}; // namespace pyroclastmpmma