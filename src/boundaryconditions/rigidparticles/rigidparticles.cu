#include "pyroclastmpm/boundaryconditions/rigidparticles/rigidparticles.cuh"

namespace pyroclastmpm
{

    extern int global_step_cpu;

    extern Real dt_cpu;

    RigidParticles::RigidParticles(const cpu_array<Vectorr> _positions,
                                   const cpu_array<int> _frames,
                                   const cpu_array<Vectorr> _locations,
                                   const cpu_array<Vectorr> _rotations,
                                   const cpu_array<OutputType> _output_formats
                                   )
    {
        output_formats = _output_formats;
        num_particles = _positions.size();

        set_default_device<Vectorr>(num_particles, _positions, positions_gpu, Vectorr::Zero());
        set_default_device<Vectorr>(num_particles, {}, velocities_gpu, Vectorr::Zero());

        launch_config.tpb = dim3(int((num_particles) / BLOCKSIZE) + 1, 1, 1);
        launch_config.bpg = dim3(BLOCKSIZE, 1, 1);

        num_frames = _frames.size();

        frames_cpu = _frames;

        locations_cpu = _locations;

        rotations_cpu = _rotations;

        // initial Center of mass
        COM = Vectorr::Zero();
        for (int pid = 0; pid < num_particles; pid++)
        {
            COM += _positions[pid];
        }
        COM /= num_particles;

        // initial translational velocity
        translational_velocity = Vectorr::Zero();

        // initial euler angles
        ROT = Vectorr::Zero();

        // initial rotation matrix
        rotation_matrix = Matrixr::Zero();
    }

    void RigidParticles::partition()
    {
        spatial.calculate_hash(positions_gpu);

        spatial.sort_hashes();

        spatial.bin_particles();
    };

    void RigidParticles::initialize(NodesContainer &nodes_ref,
                                    ParticlesContainer &particles_ref)
    {
        spatial = SpatialPartition(nodes_ref.node_start, nodes_ref.node_end,
                                   nodes_ref.node_spacing, num_particles);

        set_default_device<Vectorr>(nodes_ref.num_nodes_total, {}, normals_gpu, Vectorr::Zero());
        set_default_device<bool>(nodes_ref.num_nodes_total, {}, is_overlapping_gpu, false);
        set_default_device<int>(nodes_ref.num_nodes_total, {}, closest_rigid_particle_gpu,
                                -1);
    }

    void RigidParticles::calculate_non_rigid_grid_normals(
        NodesContainer &nodes_ref,
        ParticlesContainer &particles_ref)
    {
        KERNELS_CALC_NON_RIGID_GRID_NORMALS<<<nodes_ref.launch_config.tpb,
                                              nodes_ref.launch_config.bpg>>>(
            thrust::raw_pointer_cast(normals_gpu.data()),
            thrust::raw_pointer_cast(nodes_ref.node_ids_gpu.data()),
            thrust::raw_pointer_cast(particles_ref.dpsi_gpu.data()),
            thrust::raw_pointer_cast(particles_ref.masses_gpu.data()),
            thrust::raw_pointer_cast(particles_ref.spatial.cell_start_gpu.data()),
            thrust::raw_pointer_cast(particles_ref.spatial.cell_end_gpu.data()),
            thrust::raw_pointer_cast(particles_ref.spatial.sorted_index_gpu.data()),
            particles_ref.spatial.num_cells, particles_ref.spatial.num_cells_total);

        gpuErrchk(hipDeviceSynchronize());
    }

    void RigidParticles::calculate_overlapping_rigidbody(
        NodesContainer &nodes_ref,
        ParticlesContainer &particles_ref)
    {
        KERNEL_GET_OVERLAPPING_RIGID_BODY_GRID<<<nodes_ref.launch_config.tpb,
                                              nodes_ref.launch_config.bpg>>>(
            thrust::raw_pointer_cast(is_overlapping_gpu.data()),
            thrust::raw_pointer_cast(nodes_ref.node_ids_gpu.data()),
            thrust::raw_pointer_cast(positions_gpu.data()),
            thrust::raw_pointer_cast(spatial.bins_gpu.data()),
            particles_ref.spatial.num_cells, particles_ref.spatial.grid_start,
            particles_ref.spatial.inv_cell_size,
            particles_ref.spatial.num_cells_total, num_particles);

        gpuErrchk(hipDeviceSynchronize());
    }

    void RigidParticles::update_grid_moments(NodesContainer &nodes_ref,
                                             ParticlesContainer &particles_ref)
    {
        KERNEL_VELOCITY_CORRECTOR<<<nodes_ref.launch_config.tpb,
                                              nodes_ref.launch_config.bpg>>>(
            thrust::raw_pointer_cast(nodes_ref.moments_nt_gpu.data()),
            thrust::raw_pointer_cast(nodes_ref.moments_gpu.data()),
            thrust::raw_pointer_cast(closest_rigid_particle_gpu.data()),
            thrust::raw_pointer_cast(velocities_gpu.data()),
            thrust::raw_pointer_cast(nodes_ref.node_ids_gpu.data()),
            thrust::raw_pointer_cast(nodes_ref.masses_gpu.data()),
            thrust::raw_pointer_cast(normals_gpu.data()),
            thrust::raw_pointer_cast(is_overlapping_gpu.data()), rotation_matrix, COM,
            translational_velocity, particles_ref.spatial.grid_start,
            particles_ref.spatial.inv_cell_size,
            particles_ref.spatial.num_cells_total);
        gpuErrchk(hipDeviceSynchronize());
    }

    void RigidParticles::update_rigid_body(NodesContainer &nodes_ref,
                                           ParticlesContainer &particles_ref)

    {
        KERNEL_UPDATE_POS_RIGID<<<nodes_ref.launch_config.tpb,
                                              nodes_ref.launch_config.bpg>>>(
            thrust::raw_pointer_cast(positions_gpu.data()),
            thrust::raw_pointer_cast(velocities_gpu.data()), rotation_matrix, COM,
            translational_velocity, num_particles);
        gpuErrchk(hipDeviceSynchronize());
    }

    void RigidParticles::calculate_velocities()
    {
        const Vectorr COM_nt = locations_cpu[global_step_cpu];
        const Vectorr ROT_nt = rotations_cpu[global_step_cpu];

        translational_velocity = (COM_nt - COM) / dt_cpu;

        const Vectorr rotational_velocity = (ROT_nt - ROT);

#if DIM == 3
        const AngleAxisr rollAngle(rotational_velocity[0],
                                   Vectorr::UnitX()); // pitch

        const AngleAxisr yawAngle(rotational_velocity[1], Vectorr::UnitY()); // yaw

        const AngleAxisr pitchAngle(rotational_velocity[2],
                                    Vectorr::UnitZ()); // roll

        Quaternionr q = rollAngle * yawAngle * pitchAngle;

        // rotation_matrix = q.matrix();
        rotation_matrix = Matrixr::Zero();
#elif DIM == 2

        // const AngleAxisr rollAngle(rotational_velocity[0],Vectorr::UnitX()); // pitch

        // const AngleAxisr yawAngle(rotational_velocity[1], Vectorr::UnitY()); // yaw

        // Quaternionr q = rollAngle * yawAngle;

        // rotation_matrix = q.matrix();

#else
        rotation_matrix = Matrixr::Zero();
#endif

//           COM = COM_nt;
//           ROT = ROT_nt;
    }

    void RigidParticles::find_nearest_rigid_body(
        NodesContainer &nodes_ref,
        ParticlesContainer &particles_ref)
    {
        KERNEL_FIND_NEAREST_RIGIDPARTICLE<<<nodes_ref.launch_config.tpb,
                                              nodes_ref.launch_config.bpg>>>(
            thrust::raw_pointer_cast(closest_rigid_particle_gpu.data()),
            thrust::raw_pointer_cast(positions_gpu.data()),
            thrust::raw_pointer_cast(nodes_ref.node_ids_gpu.data()),
            thrust::raw_pointer_cast(nodes_ref.masses_gpu.data()),
            thrust::raw_pointer_cast(spatial.cell_start_gpu.data()),
            thrust::raw_pointer_cast(spatial.cell_end_gpu.data()),
            thrust::raw_pointer_cast(spatial.sorted_index_gpu.data()),
            thrust::raw_pointer_cast(is_overlapping_gpu.data()), spatial.num_cells,
            spatial.grid_start, spatial.inv_cell_size, spatial.num_cells_total);

        gpuErrchk(hipDeviceSynchronize());
    };

    void RigidParticles::apply_on_nodes_moments(NodesContainer &nodes_ref,
                                                ParticlesContainer &particles_ref){
          // TODO move this somewhere else?
          if (global_step_cpu == 0) {
            initialize(nodes_ref, particles_ref);
          }

          partition();

          if (num_frames > 0) {
            calculate_velocities();  // host bound
          }

          calculate_overlapping_rigidbody(nodes_ref, particles_ref);

          calculate_non_rigid_grid_normals(nodes_ref, particles_ref);

          find_nearest_rigid_body(nodes_ref, particles_ref);

          if (num_frames > 0) {
            update_rigid_body(nodes_ref, particles_ref);
          }

          // std::cout << rotation_matrix << "\n -----------------------------\n";
          update_grid_moments(nodes_ref, particles_ref);
    };

    void RigidParticles::output_vtk()
    {
          vtkSmartPointer<vtkPolyData> polydata = vtkSmartPointer<vtkPolyData>::New();

          set_vtk_points(positions_gpu, polydata);

          cpu_array<Vectorr> velocities_cpu = velocities_gpu;
          set_vtk_pointdata<Vectorr>(velocities_cpu, polydata, "velocities");

        //   // cpu_array<Vector3r> velocities_cpu = velocities_gpu;
        //   // set_vtk_pointdata(velocities_cpu, polydata, "Velocity");

        //   vtkSmartPointer<vtkTransform> transform =
        //   vtkSmartPointer<vtkTransform>::New();

        //   // transform->Translate(0.575, 0.4  , 0.);
        //   // transform->RotateZ(90.0);
        //   // transform->Translate(-0.575, -0.4  , -0.);

        //   // vtkSmartPointer<vtkTransformPolyDataFilter> transformFilter =
        //   vtkSmartPointer<vtkTransformPolyDataFilter>::New();
        //   // transformFilter->SetInputData(polydata);
        //   // transformFilter->SetTransform(transform);
        //   // transformFilter->Update();

        //    vtkSmartPointer<vtkPolyData> polydata_transformed =
        //    vtkSmartPointer<vtkPolyData>::New();

        //   //  polydata_transformed = transformFilter->GetOutput();
        //   // transformFilter->

        //   write_vtk_polydata(polydata, "rigidbody");

        for (auto format : output_formats)
        {
            write_vtk_polydata(polydata, "rigidbody", format);
        }
    }

} // namespace pyroclastmpm