
#include "pyroclastmpm/solver/musl/musl.cuh"

namespace pyroclastmpm
{

  MUSL::MUSL(
      ParticlesContainer _particles,
      NodesContainer _nodes,
      cpu_array<MaterialType> _materials,
      cpu_array<BoundaryConditionType> _boundaryconditions,
      Real _alpha) : USL(_particles, _nodes, _materials, _boundaryconditions, _alpha)
  {
  }

  void MUSL::solve()
  {
    reset(); // inherited from solver class

    particles.partition();

    calculate_shape_function(); // inherited form solver class

   for (int bc_id = 0; bc_id < boundaryconditions.size(); bc_id++)
    {
      std::visit([&](auto &arg)
                 { arg.apply_on_particles(particles); },
                 boundaryconditions[bc_id]);
    }
    
    P2G(); // inherited from usl class

    for (int bc_id = 0; bc_id < boundaryconditions.size(); bc_id++)
    {
      std::visit([&](auto &arg)
                 { arg.apply_on_nodes_f_ext(nodes); },
                 boundaryconditions[bc_id]);
    }

    nodes.integrate();

    for (int bc_id = 0; bc_id < boundaryconditions.size(); bc_id++)
    {
      std::visit([&](auto &arg)
                 { arg.apply_on_nodes_moments(nodes, particles); },
                 boundaryconditions[bc_id]);
    }

    G2P_double_mapping();

    P2G_double_mapping();

    for (int bc_id = 0; bc_id < boundaryconditions.size(); bc_id++)
    {
      std::visit([&](auto &arg)
                 { arg.apply_on_nodes_moments(nodes, particles); },
                 boundaryconditions[bc_id]);
    }

    G2P();

    stress_update(); // inherited from solver class

    for (int bc_id = 0; bc_id < boundaryconditions.size(); bc_id++)
    {
      std::visit([&](auto &arg)
                 { arg.apply_on_particles(particles); },
                 boundaryconditions[bc_id]);
    }
  }

  void MUSL::P2G_double_mapping()
  {

    KERNEL_MUSL_P2G_DOUBLE_MAPPING<<<nodes.launch_config.tpb,
                                     nodes.launch_config.bpg>>>(
        thrust::raw_pointer_cast(nodes.moments_nt_gpu.data()),
        thrust::raw_pointer_cast(nodes.masses_gpu.data()),
        thrust::raw_pointer_cast(nodes.node_ids_gpu.data()),
        thrust::raw_pointer_cast(particles.velocities_gpu.data()),
        thrust::raw_pointer_cast(particles.dpsi_gpu.data()),
        thrust::raw_pointer_cast(particles.psi_gpu.data()),
        thrust::raw_pointer_cast(particles.masses_gpu.data()),
        thrust::raw_pointer_cast(particles.spatial.cell_start_gpu.data()),
        thrust::raw_pointer_cast(particles.spatial.cell_end_gpu.data()),
        thrust::raw_pointer_cast(particles.spatial.sorted_index_gpu.data()),
        nodes.num_nodes, nodes.num_nodes_total);

    gpuErrchk(hipDeviceSynchronize());
  }

  void MUSL::G2P_double_mapping()
  {

    KERNEL_MUSL_G2P_DOUBLE_MAPPING<<<particles.launch_config.tpb,
                                     particles.launch_config.bpg>>>(
        thrust::raw_pointer_cast(particles.velocities_gpu.data()),
        thrust::raw_pointer_cast(particles.positions_gpu.data()),
        thrust::raw_pointer_cast(particles.dpsi_gpu.data()),
        // thrust::raw_pointer_cast(nodes.node_ids_gpu.data()),
        thrust::raw_pointer_cast(particles.spatial.bins_gpu.data()),
        thrust::raw_pointer_cast(particles.psi_gpu.data()),
        thrust::raw_pointer_cast(nodes.moments_gpu.data()),
        thrust::raw_pointer_cast(nodes.moments_nt_gpu.data()),
        thrust::raw_pointer_cast(nodes.masses_gpu.data()),
        particles.spatial.num_cells,
        particles.num_particles, alpha);
    gpuErrchk(hipDeviceSynchronize());
  };

  void MUSL::G2P()
  {
    KERNEL_MUSL_G2P<<<particles.launch_config.tpb,
                      particles.launch_config.bpg>>>(
        thrust::raw_pointer_cast(particles.velocity_gradient_gpu.data()),
        thrust::raw_pointer_cast(particles.F_gpu.data()),
        thrust::raw_pointer_cast(particles.volumes_gpu.data()),
        thrust::raw_pointer_cast(nodes.moments_nt_gpu.data()),
        thrust::raw_pointer_cast(particles.dpsi_gpu.data()),
        thrust::raw_pointer_cast(particles.spatial.bins_gpu.data()),
        thrust::raw_pointer_cast(particles.volumes_original_gpu.data()),
        thrust::raw_pointer_cast(particles.psi_gpu.data()),
        thrust::raw_pointer_cast(particles.masses_gpu.data()),
        thrust::raw_pointer_cast(nodes.masses_gpu.data()),
        particles.spatial.num_cells,
        particles.num_particles);

    gpuErrchk(hipDeviceSynchronize());
  };

} // namespace pyroclastmpm